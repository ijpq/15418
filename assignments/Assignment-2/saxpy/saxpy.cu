#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);

__global__ void saxpy_kernel(int N, float alpha, float *x, float *y, float *result) {
    // Compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
        result[index] = alpha * x[index] + y[index];
}

void saxpyCuda(int N, float alpha, float *xarray, float *yarray, float *resultarray) {
    int totalBytes = sizeof(float) * 3 * N;

    // Compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float *device_x;
    float *device_y;
    float *device_result;

    //
    // TODO allocate device memory buffers on the GPU using hipMalloc
    //

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO copy input arrays to the GPU using hipMemcpy
    //

    // run kernel
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();

    //
    // TODO copy result from GPU using hipMemcpy
    //

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // TODO free memory buffers on the GPU
}

void printCudaInfo() {
    // For fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
