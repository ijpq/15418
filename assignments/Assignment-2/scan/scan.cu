#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);

/* Helper function to round up to a power of 2.
 */
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

void exclusive_scan(int *device_data, int length) {
    /* TODO
     * Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the data in device memory
     * The data are initialized to the inputs.  Your code should
     * do an in-place scan, generating the results in the same array.
     * This is host code -- you will need to declare one or more CUDA
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the data array is sized to accommodate the next
     * power of 2 larger than the input.
     */
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int *inarray, int *end, int *resultarray) {
    int *device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int *inarray, int *end, int *resultarray) {
    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */
    return 0;
}

/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}

void printCudaInfo() {
    // For fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
