#include "hip/hip_runtime.h"
#include <algorithm>
#include <string>
#define _USE_MATH_DEFINES
#define THREADS_PER_BLOCK 16
#define PIXEL_GRID_DIM 64
#define SCAN_BLOCK_DIM 16*16
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"
#include "renderAlongWithPixel.h"
#include "exclusiveScan.cu_inl"
#include "hipError_t.cu_inl"
#include <cstdint>

#include <vector>
#include <iostream>
#include <ctime>


#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess) {\
        printf("Error: %s:%d, ", __FILE__, __LINE__);\
        printf("code:%d, reason: %s\n",error, hipGetErrorString(error));\
        exit(-1);\
    }\
}
////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// This stores the global constants
struct GlobalConstants {

    SceneName sceneName;

    int numberOfCircles;

    float *position;
    float *velocity;
    float *color;
    float *radius;

    int imageWidth;
    int imageHeight;
    float *imageData;
};

// Global variable that is in scope, but read-only, for all CUDA
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (We didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// Read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int cuConstNoiseYPermutationTable[256];
__constant__ int cuConstNoiseXPermutationTable[256];
__constant__ float cuConstNoise1DValueTable[256];

// Color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float cuConstColorRamp[COLOR_MAP_SIZE][3];

// Include parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "lookupColor.cu_inl"
#include "noiseCuda.cu_inl"
using namespace std;


////////////////////////////////////////////////////////////////////////////////////////


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height - imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
//
// Update positions of fireworks
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = M_PI;
    const float maxDist = 0.25f;

    float *velocity = cuConstRendererParams.velocity;
    float *position = cuConstRendererParams.position;
    float *radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update
        return;
    }

    // Determine the firework center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i + 1];

    // Update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j + 1] += velocity[index3j + 1] * dt;

    // Firework sparks
    float sx = position[index3j];
    float sy = position[index3j + 1];

    // Compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // Compute distance from fire-work
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position
        // Random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi) / NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j + 1] = position[index3i + 1] + y;
        position[index3j + 2] = 0.0f;

        // Travel scaled unit length
        velocity[index3j] = cosA / 5.0;
        velocity[index3j + 1] = sinA / 5.0;
        velocity[index3j + 2] = 0.0f;
    }
}

// kernelAdvanceHypnosis
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    float *radius = cuConstRendererParams.radius;

    float cutOff = 0.5f;
    // Place circle back in center after reaching threshold radisus
    if (radius[index] > cutOff) {
        radius[index] = 0.02f;
    } else {
        radius[index] += 0.01f;
    }
}

// kernelAdvanceBouncingBalls
//
// Update the position of the balls
__global__ void kernelAdvanceBouncingBalls() {
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    float *velocity = cuConstRendererParams.velocity;
    float *position = cuConstRendererParams.position;

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3 + 1];
    float oldPosition = position[index3 + 1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition
        return;
    }

    if (position[index3 + 1] < 0 && oldVelocity < 0.f) { // bounce ball
        velocity[index3 + 1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3 + 1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3 + 1] += velocity[index3 + 1] * dt;

    if (fabsf(velocity[index3 + 1] - oldVelocity) < epsilon && oldPosition < 0.0f &&
        fabsf(position[index3 + 1] - oldPosition) < epsilon) { // stop ball
        velocity[index3 + 1] = 0.f;
        position[index3 + 1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// Move the snowflake animation forward one time step.  Update circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float *positionPtr = &cuConstRendererParams.position[index3];
    float *velocityPtr = &cuConstRendererParams.velocity[index3];

    // Load from global memory
    float3 position = *((float3 *)positionPtr);
    float3 velocity = *((float3 *)velocityPtr);

    // Hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // Add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // Drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // Update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // Update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // If the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ((position.y + radius < 0.f) || (position.x + radius) < -0.f || (position.x - radius) > 1.f) {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // Restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // Store updated positions and velocities to global memory
    *((float3 *)positionPtr) = position;
    *((float3 *)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// Given a pixel and a circle, determine the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void shadePixel(float2 pixelCenter, float3 p, float4 *imagePtr, int circleIndex) {
    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];
    float maxDist = rad * rad;

    // Circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // There is a non-zero contribution.  Now compute the shading value

    // Suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks, etc., to implement the conditional.  It
    // would be wise to perform this logic outside of the loops in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES ||
        cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f - p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // Simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3 *)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // Global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void kernelRenderCircles() {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    int index3 = 3 * index;

    // Read position and radius
    float3 p = *(float3 *)(&cuConstRendererParams.position[index3]);
    float rad = cuConstRendererParams.radius[index];

    // Compute the bounding box of the circle. The bound is in integer
    // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // A bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    // For all pixels in the bounding box
    for (int pixelY = screenMinY; pixelY < screenMaxY; pixelY++) {
        float4 *imgPtr = (float4 *)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]);
        for (int pixelX = screenMinX; pixelX < screenMaxX; pixelX++) {
            float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
            shadePixel(pixelCenterNorm, p, imgPtr, index);
            imgPtr++;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer() {
    image = NULL;

    numberOfCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {
    if (image) {
        delete image;
    }

    if (position) {
        delete[] position;
        delete[] velocity;
        delete[] color;
        delete[] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image *CudaRenderer::getImage() {
    // Need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data, cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numberOfCircles, position, velocity, color, radius);
}

void CudaRenderer::setup() {
    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce RTX 2080") == 0) {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU) {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numberOfCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numberOfCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numberOfCircles = numberOfCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // Also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int *permX;
    int *permY;
    float *value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // Copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f}, {1.f, 1.f, 1.f}, {.8f, .9f, 1.f}, {.8f, .9f, 1.f}, {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void CudaRenderer::allocOutputImage(int width, int height) {
    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x,
                 (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void CudaRenderer::advanceAnimation() {
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numberOfCircles + blockDim.x - 1) / blockDim.x);

    // Only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) {
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
    }
    hipDeviceSynchronize();
}


__device__ __inline__ int isWithinCircle(const float2 &pixelCenterNorm, const float3 &circlePos) {
    float diffX = pixelCenterNorm.x - circlePos.x;
    float diffY = pixelCenterNorm.y - circlePos.y;
    float pixelDist = diffX * diffX + diffY * diffY;
    float maxDist = circlePos.z * circlePos.z;
    return pixelDist <= maxDist ? 1 : 0;
}

__device__ void myShadePixel(const int &circleIndex, const float2 &pixelCenterNorm, float4 *imagePtr) {
    float3 rgb;
    float alpha; 
    int index3 = circleIndex * 3;

    // TODO: template
    if (cuConstRendererParams.sceneName == SNOWFLAKES || \
        cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;
        float rad = cuConstRendererParams.radius[circleIndex];
        float diffX = pixelCenterNorm.x - cuConstRendererParams.position[index3];
        float diffY = pixelCenterNorm.y - cuConstRendererParams.position[index3 + 1];
        float pixelDist = diffX * diffX + diffY * diffY;
        
        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f - cuConstRendererParams.position[index3 + 2]);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f);
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {

        rgb = *(float3 *)&(cuConstRendererParams.color[index3]);
        alpha = .5f;

    }
    
    float oneMinusAlpha = 1.f - alpha;

    // FIXME: atomic if this device code rendering more than one pixel.

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    *imagePtr = newColor;


    return ;
}
__device__
void updateRGBA(const SceneName &sceneName, const int &circleIndex, const int &indexX, const int &indexY, float4 &imageRGBA) {
    int index3 = 3 * circleIndex;
    float2 circleXY = make_float2(cuConstRendererParams.position[index3], cuConstRendererParams.position[index3+1]);
    float2 pixelCenterXY = \
        make_float2( \
           (1.f / cuConstRendererParams.imageWidth) * \
           (static_cast<float>(indexX) + .5f), \
           (1.f / cuConstRendererParams.imageHeight) * \
           (static_cast<float>(indexY) + .5f) \
        );
    float diffX = circleXY.x - pixelCenterXY.x;
    float diffY = circleXY.y - pixelCenterXY.y;
    float pixelDist = diffX * diffX + diffY * diffY;
    float rad = cuConstRendererParams.radius[circleIndex];
    if (pixelDist > rad*rad) {
        return;
    }

    float3 rgb;
    float alpha;
    if (sceneName == SNOWFLAKES_SINGLE_FRAME || sceneName == SNOWFLAKES) {
        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;
        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f - cuConstRendererParams.position[index3 + 2]);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f);
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);
    } else {
        rgb = *(float3 *)&cuConstRendererParams.color[index3];
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;
    float4 newRGBA;
    newRGBA.x = alpha * rgb.x + oneMinusAlpha * imageRGBA.x;
    newRGBA.y = alpha * rgb.y + oneMinusAlpha * imageRGBA.y;
    newRGBA.z = alpha * rgb.z + oneMinusAlpha * imageRGBA.z;
    newRGBA.w = alpha + imageRGBA.w;
    imageRGBA = newRGBA;
    return ;
}

__device__
void myRenderPixel(const float4 &imageRGBA, const int &indexX, const int &indexY, const int &imageWidth) {
    *(float4 *)(&cuConstRendererParams.imageData[4 * (indexY * imageWidth + indexX)]) = imageRGBA;
    return ;
}

__global__ void kernelRenderPixels() {
    // 把线程映射到像素点，边界块中的多余线程可能不扔掉更好
    int indexX = threadIdx.x + blockDim.x * blockIdx.x;
    int indexY = threadIdx.y + blockDim.y * blockIdx.y;
    int imageWidth = cuConstRendererParams.imageWidth;
    int imageHeight = cuConstRendererParams.imageHeight;
    if (indexX >= imageWidth || indexY >= imageHeight) {
        return ;
    }

    // 计算画布格边界
    int gridMinX = blockIdx.x * THREADS_PER_BLOCK;
    int gridMinY = blockIdx.y * THREADS_PER_BLOCK;
    int gridMaxX = min(gridMinX + THREADS_PER_BLOCK, imageWidth);
    int gridMaxY = min(gridMinY + THREADS_PER_BLOCK, imageHeight);

    // 设定块内线程，计算圆数
    int localTidx = threadIdx.y * blockDim.y + threadIdx.x;
    int circleRange = (cuConstRendererParams.numberOfCircles + THREADS_PER_BLOCK * THREADS_PER_BLOCK - 1) / (THREADS_PER_BLOCK * THREADS_PER_BLOCK);
    int firstCircleIndex = localTidx * circleRange;
    int lastCircleIndex = min(firstCircleIndex + circleRange, cuConstRendererParams.numberOfCircles);

    // 当前线程负责的range中有多少个圆出现在的对应的grid中
    uint nrCircleThread = 0;

    // 圆id很大 32t
    uint *cirIdxs = nullptr;
    if (hipError_t errCode = hipMalloc(&cirIdxs, sizeof(uint) * circleRange)) {
        printf("cuda malloc error\n");
        return ;
    }

    for (int cidx = firstCircleIndex; cidx < lastCircleIndex; cidx++) {
        int index3 = cidx * 3;
        float3 circlePos = make_float3( \
            cuConstRendererParams.position[index3], \
            cuConstRendererParams.position[index3+1], \
            cuConstRendererParams.radius[cidx]);
        short circleMinX = static_cast<short>(imageWidth * (circlePos.x - circlePos.z));
        short circleMaxX = static_cast<short>(imageWidth * (circlePos.x + circlePos.z)) + 1; 
        short circleMinY = static_cast<short>(imageHeight * (circlePos.y - circlePos.z));
        short circleMaxY = static_cast<short>(imageHeight * (circlePos.y + circlePos.z)) + 1;
        if (circleMinX >= gridMaxX || \
            circleMaxX < gridMinX || \
            circleMinY >= gridMaxY || \
            circleMaxY < gridMinY) {
            continue;
        }
        // arr记录圆id
        cirIdxs[nrCircleThread++] = cidx;
    }

    // parallel prefix sum
    // 每个线程负责圆数量
    __shared__ uint tidxNumCirs[THREADS_PER_BLOCK * THREADS_PER_BLOCK];
    // 前缀和后，画布格里可能包含全部圆，32t
    __shared__ uint tidxNumCirsPrefixSum[THREADS_PER_BLOCK * THREADS_PER_BLOCK];
    __shared__ uint prefixSumArr[THREADS_PER_BLOCK * THREADS_PER_BLOCK * 2];
    // arr记录圆数量
    tidxNumCirs[localTidx] = nrCircleThread;
    // 块内同步，也是画布格同步，准备前缀和数组
    __syncthreads();
    sharedMemExclusiveScan(localTidx, tidxNumCirs, tidxNumCirsPrefixSum, prefixSumArr, THREADS_PER_BLOCK*THREADS_PER_BLOCK);
    // 同步，取得前缀和数组
    __syncthreads();
    

    // 画布格内圆的数量决定了用多长的数组存圆的idx
    int nrCirGrid = 0;
    if (localTidx == THREADS_PER_BLOCK * THREADS_PER_BLOCK - 1) {
        nrCirGrid = tidxNumCirsPrefixSum[localTidx] + nrCircleThread;
    }
    __syncthreads();
    // 画布格内圆id
    __shared__ uint gridCirIdx[100*1000];
    for (int i = 0; i < nrCircleThread; i++) {
        gridCirIdx[i + tidxNumCirsPrefixSum[localTidx]] = cirIdxs[i];
    }
    __syncthreads();
    hipFree(cirIdxs);
    
    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;
    float2 pixelCenterNorm = \
        make_float2( \
            invWidth * (static_cast<float>(indexX) + .5f), \
            invHeight * (static_cast<float>(indexY) + .5f) \
            );
    for (int circleIndex = 0; circleIndex < nrCirGrid; circleIndex++) {
        float3 circlePos = make_float3( \
            cuConstRendererParams.position[3 * gridCirIdx[circleIndex]], \
            cuConstRendererParams.position[3 * gridCirIdx[circleIndex] + 1], \
            cuConstRendererParams.radius[gridCirIdx[circleIndex]]);
        if (isWithinCircle(pixelCenterNorm, circlePos)) {
            float4 *imagePtr = (float4 *)(&cuConstRendererParams.imageData[4 * (indexY * imageWidth + indexX)]);
            myShadePixel(gridCirIdx[circleIndex], pixelCenterNorm, imagePtr);
        }
    }


    return ;
}

void CudaRenderer::doRenderPixels() {
    int imageWidth = image->width;
    int imageHeight = image->height;
    dim3 blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 gridDim((imageWidth + THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK, (imageHeight + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    kernelRenderPixels<<<gridDim, blockDim>>>();
    CHECK(hipDeviceSynchronize());
}

void CudaRenderer::render() {
    // 256 threads per block is a healthy number

    // doRenderCircles();
    
    doRenderPixels();

    hipDeviceSynchronize();
}
