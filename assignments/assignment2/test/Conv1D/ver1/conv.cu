#include "hip/hip_runtime.h"
#define SRC_SIZE 10
#define THREADS_PER_BLOCK 256
#define DST_SIZE (SRC_SIZE - 2)
#include <iostream>
#include <time.h>
#include "../../parallelreduce/common/common.h"

using namespace std;
__global__ void conv(float *src, float *dst, size_t dst_N) {

    // mapping threads to dst data
    size_t thd = threadIdx.x + blockDim.x * blockIdx.x;
    if (thd >= dst_N) return ; 
    dst[thd] = (src[thd] + src[thd+1] + src[thd+2]) / 3.f;
    return ;
}

void serial(float *src, float *dst, const int &dst_N) {
    for (int i = 0; i < dst_N; i++) {
        dst[i] = (src[i] + src[i+1] + src[i+2]) / 3.f;
    }
    return ;
}

void output_dst(float *dst, size_t size) {
    for (auto i =0; i < size; i++) {
        cout << dst[i] << ",";
    }
    cout << endl;
    return ;
}


int main(void) {
    
    size_t src_alloc = sizeof(float) * SRC_SIZE;
    size_t dst_alloc = sizeof(float) * DST_SIZE;
    float *src = (float *)malloc(src_alloc);
    float *dst = (float *)malloc(dst_alloc);
    memset(src, 0, src_alloc);
    
    for (auto i =0; i < SRC_SIZE; i++) {
        src[i] = i+0.1f;
    }
    clock_t start = seconds();
    serial(src, dst, DST_SIZE);
    cout << ((double)(seconds()-start))   << endl;
    cout << "from cpu" << endl;
    output_dst(dst, DST_SIZE);

    float *d_src = nullptr;
    float *d_dst = nullptr;
    hipMalloc(&d_src, src_alloc);
    hipMalloc(&d_dst, dst_alloc);
    hipMemcpy(d_src, src, src_alloc, hipMemcpyHostToDevice);

    memset(dst, 0, dst_alloc);
    int num_threads = THREADS_PER_BLOCK;
    int num_blocks = (DST_SIZE + num_threads -1) / num_threads; 
    dim3 blocks(num_blocks);
    dim3 threads(num_threads);
    start = seconds();
    conv<<<blocks, threads>>>(d_src, d_dst, DST_SIZE);
    hipDeviceSynchronize();
    cout << ((double)(seconds()-start))  << endl;
    hipMemcpy(dst, d_dst, dst_alloc, hipMemcpyDeviceToHost);
    cout << "from gpu" << endl;
    output_dst(dst, DST_SIZE);
    
    return 0;
}
